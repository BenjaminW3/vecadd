
#include <hip/hip_runtime.h>
//-----------------------------------------------------------------------------
//! \file
//! Copyright 2015 Benjamin Worpitz
//!
//! This file is part of vecadd.
//!
//! vecadd is free software: you can redistribute it and/or modify
//! it under the terms of the GNU Lesser General Public License as published by
//! the Free Software Foundation, either version 3 of the License, or
//! (at your option) any later version.
//!
//! vecadd is distributed in the hope that it will be useful,
//! but WITHOUT ANY WARRANTY; without even the implied warranty of
//! MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See the
//! GNU Lesser General Public License for more details.
//!
//! You should have received a copy of the GNU Lesser General Public License
//! along with vecadd.
//! If not, see <http://www.gnu.org/licenses/>.
//-----------------------------------------------------------------------------

#if defined(VECADD_BUILD_PAR_CUDA)

    #include <vecadd/par/Cuda.h>

    #include <vecadd/common/Time.h> // getTimeSec

    #include <cuda_runtime.h>

    #include <stdio.h>              // printf
    #include <math.h>               // ceil
    #include <algorithm>            // std::min

    #define VECADD_CUDA_RT_CHECK(cmd) {cudaError_t error = cmd; if(error!=cudaSuccess){printf("<%s>:%i ",__FILE__,__LINE__); printf("[CUDA] Error: %s\n", cudaGetErrorString(error));}}

        //-----------------------------------------------------------------------------
        // This function only works for square blocks.
        //-----------------------------------------------------------------------------
        __global__ void vecadd_axpy_par_cuda_kernel(
            TSize const n,
            TElem const alpha,
            TElem const * const VECADD_RESTRICT X,
            TElem * const VECADD_RESTRICT Y)
        {
            TSize const i(blockIdx.x*blockDim.x + threadIdx.x);

            if(i < n)
            {
                Y[i] = alpha * X[i] + Y[i];
            }
        }
        //-----------------------------------------------------------------------------
        //
        //-----------------------------------------------------------------------------
        TReturn vecadd_axpy_par_cuda(
            TSize const n,
            TElem const alpha,
            TElem const * const VECADD_RESTRICT X,
            TElem * const VECADD_RESTRICT Y)
        {
            // VECADD_CUDA_RT_CHECK(cudaSetDevice(0));
            cudaStream_t stream;
            VECADD_CUDA_RT_CHECK(cudaStreamCreate(&stream));

            // Get its properties.
            cudaDeviceProp cudaDevProp;
            VECADD_CUDA_RT_CHECK(cudaGetDeviceProperties(
                &cudaDevProp,
                0));

            TSize gridThreadExtent[] = {n};
            TSize blockThreadExtent[] = {cudaDevProp.maxThreadsDim[0]};

            // Restrict the max block thread extent with the grid thread extent.
            // This removes dimensions not required in the given grid thread extent.
            // This has to be done before the maxThreadsPerBlock clipping to get the maximum correctly.
            for(TSize i(0); i<1; ++i)
            {
                blockThreadExtent[i] = std::min(blockThreadExtent[i], gridThreadExtent[i]);
            }

            // Restrict it to its minimum component.
            // For example (512, 256) will get (256, 256).
            auto minBlockThreadExtent(blockThreadExtent[0]);
            for(TSize i(1); i<1; ++i)
            {
                minBlockThreadExtent = std::min(minBlockThreadExtent, blockThreadExtent[i]);
            }
            for(TSize i(0); i<1; ++i)
            {
                blockThreadExtent[i] = minBlockThreadExtent;
            }

            // Adjust blockThreadExtent if its product is too large.
            if ((blockThreadExtent[0]) > cudaDevProp.maxThreadsPerBlock)
            {
                // Satisfy the following equation:
                // udaDevProp.maxThreadsPerBlock >= blockThreadExtent[0]*blockThreadExtent[1]
                // For example 1024 >= 512 * 512

                // For equal block thread extent this is easily the nth root of cudaDevProp.maxThreadsPerBlock.
                double const fNthRoot(std::pow(cudaDevProp.maxThreadsPerBlock, 1.0 / 1.0));
                auto const nthRoot(static_cast<TSize>(fNthRoot));
                for(TSize i(0); i<1; ++i)
                {
                    blockThreadExtent[i] = nthRoot;
                }
            }

            // Set the grid block extent (rounded to the next integer not less then the quotient.
            TSize gridBlockExtent[] = {1};
            for(TSize i(0); i<1; ++i)
            {
                gridBlockExtent[i] =
                    static_cast<TSize>(
                        std::ceil(static_cast<double>(gridThreadExtent[i])
                            / static_cast<double>(blockThreadExtent[i])));
            }

            dim3 const dimBlock(blockThreadExtent[0]);
            dim3 const dimGrid(gridBlockExtent[0]);

            VECADD_TIME_START;

            vecadd_axpy_par_cuda_kernel<<<
                dimGrid,
                dimBlock,
                0,
                stream>>>(
                    n,
                    alpha,
                    X,
                    Y);

            VECADD_CUDA_RT_CHECK(cudaStreamSynchronize(stream));

            VECADD_TIME_END;

            VECADD_CUDA_RT_CHECK(cudaStreamDestroy(stream));

            VECADD_TIME_RETURN;
        }
#endif
