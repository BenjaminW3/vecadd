
#include <hip/hip_runtime.h>
//-----------------------------------------------------------------------------
//! \file
//! Copyright 2015 Benjamin Worpitz
//!
//! This file is part of vecadd.
//!
//! vecadd is free software: you can redistribute it and/or modify
//! it under the terms of the GNU Lesser General Public License as published by
//! the Free Software Foundation, either version 3 of the License, or
//! (at your option) any later version.
//!
//! vecadd is distributed in the hope that it will be useful,
//! but WITHOUT ANY WARRANTY; without even the implied warranty of
//! MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See the
//! GNU Lesser General Public License for more details.
//!
//! You should have received a copy of the GNU Lesser General Public License
//! along with vecadd.
//! If not, see <http://www.gnu.org/licenses/>.
//-----------------------------------------------------------------------------

#if defined(VECADD_BUILD_PAR_CUDA)

    #include <vecadd/par/Cuda.h>

    #include <vecadd/common/Time.h> // getTimeSec

    #include <cuda_runtime.h>

    #include <stdio.h>              // printf
    #include <math.h>               // ceil
    #include <algorithm>            // std::min

    #define VECADD_CUDA_RT_CHECK(cmd) {cudaError_t error = cmd; if(error!=cudaSuccess){printf("<%s>:%i ",__FILE__,__LINE__); printf("[CUDA] Error: %s\n", cudaGetErrorString(error));}}

        //-----------------------------------------------------------------------------
        // This function only works for square blocks.
        //-----------------------------------------------------------------------------
        __global__ void vecadd_axpy_par_cuda_kernel(
            TSize const n,
            TElem const alpha,
            TElem const * const VECADD_RESTRICT X,
            TElem * const VECADD_RESTRICT Y)
        {
            TSize const i(blockIdx.x*blockDim.x + threadIdx.x);

            if(i < n)
            {
                Y[i] = alpha * X[i] + Y[i];
            }
        }
        //-----------------------------------------------------------------------------
        //
        //-----------------------------------------------------------------------------
        TReturn vecadd_axpy_par_cuda(
            TSize const n,
            TElem const alpha,
            TElem const * const VECADD_RESTRICT X,
            TElem * const VECADD_RESTRICT Y)
        {
            // VECADD_CUDA_RT_CHECK(cudaSetDevice(0));
            cudaStream_t stream;
            VECADD_CUDA_RT_CHECK(cudaStreamCreate(&stream));

            // Get its properties.
            cudaDeviceProp cudaDevProp;
            VECADD_CUDA_RT_CHECK(cudaGetDeviceProperties(
                &cudaDevProp,
                0));

            TSize gridThreadExtents[] = {n};
            TSize blockThreadExtents[] = {cudaDevProp.maxThreadsDim[0]};

            // Restrict the max block thread extents with the grid thread extents.
            // This removes dimensions not required in the given grid thread extents.
            // This has to be done before the maxThreadsPerBlock clipping to get the maximum correctly.
            for(TSize i(0); i<1; ++i)
            {
                blockThreadExtents[i] = std::min(blockThreadExtents[i], gridThreadExtents[i]);
            }

            // Restrict it to its minimum component.
            // For example (512, 256) will get (256, 256).
            auto minBlockThreadExtent(blockThreadExtents[0]);
            for(TSize i(1); i<1; ++i)
            {
                minBlockThreadExtent = std::min(minBlockThreadExtent, blockThreadExtents[i]);
            }
            for(TSize i(0); i<1; ++i)
            {
                blockThreadExtents[i] = minBlockThreadExtent;
            }

            // Adjust blockThreadExtents if its product is too large.
            if ((blockThreadExtents[0]) > cudaDevProp.maxThreadsPerBlock)
            {
                // Satisfy the following equation:
                // udaDevProp.maxThreadsPerBlock >= blockThreadExtents[0]*blockThreadExtents[1]
                // For example 1024 >= 512 * 512

                // For equal block thread extent this is easily the nth root of cudaDevProp.maxThreadsPerBlock.
                double const fNthRoot(std::pow(cudaDevProp.maxThreadsPerBlock, 1.0 / 1.0));
                auto const nthRoot(static_cast<TSize>(fNthRoot));
                for(TSize i(0); i<1; ++i)
                {
                    blockThreadExtents[i] = nthRoot;
                }
            }

            // Set the grid block extents (rounded to the next integer not less then the quotient.
            TSize gridBlockExtents[] = {1};
            for(TSize i(0); i<1; ++i)
            {
                gridBlockExtents[i] =
                    static_cast<TSize>(
                        std::ceil(static_cast<double>(gridThreadExtents[i])
                            / static_cast<double>(blockThreadExtents[i])));
            }

            dim3 const dimBlock(blockThreadExtents[0]);
            dim3 const dimGrid(gridBlockExtents[0]);

            VECADD_TIME_START;

            vecadd_axpy_par_cuda_kernel<<<
                dimGrid,
                dimBlock,
                0,
                stream>>>(
                    n,
                    alpha,
                    X,
                    Y);

            VECADD_CUDA_RT_CHECK(cudaStreamSynchronize(stream));

            VECADD_TIME_END;

            VECADD_CUDA_RT_CHECK(cudaStreamDestroy(stream));

            VECADD_TIME_RETURN;
        }
#endif
