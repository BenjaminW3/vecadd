
#include <hip/hip_runtime.h>
//-----------------------------------------------------------------------------
//! \file
//! Copyright 2015 Benjamin Worpitz
//!
//! This file is part of vecadd.
//!
//! vecadd is free software: you can redistribute it and/or modify
//! it under the terms of the GNU Lesser General Public License as published by
//! the Free Software Foundation, either version 3 of the License, or
//! (at your option) any later version.
//!
//! vecadd is distributed in the hope that it will be useful,
//! but WITHOUT ANY WARRANTY; without even the implied warranty of
//! MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See the
//! GNU Lesser General Public License for more details.
//!
//! You should have received a copy of the GNU Lesser General Public License
//! along with vecadd.
//! If not, see <http://www.gnu.org/licenses/>.
//-----------------------------------------------------------------------------

#if defined(VECADD_BUILD_PAR_ALPAKA_ACC_GPU_CUDA)

    #include <vecadd/par/Alpaka.h>

    #include <vecadd/par/Alpaka.hpp>

    //-----------------------------------------------------------------------------
    //
    //-----------------------------------------------------------------------------
    TReturn vecadd_axpy_par_alpaka_gpu_cuda(
        TSize const n,
        TElem const alpha,
        TElem const * const X,
        TElem * const Y)
    {
        return
            vecadd_axpy_par_alpaka<alpaka::acc::AccGpuCudaRt<alpaka::dim::DimInt<1u>, TSize>, AxpyAlpakaKernel>(
                n,
                1u,
                alpha,
                X,
                Y);
    }
    //-----------------------------------------------------------------------------
    //
    //-----------------------------------------------------------------------------
    TReturn vecadd_axpy_par_vec2_alpaka_gpu_cuda(
        TSize const n,
        TElem const alpha,
        TElem const * const X,
        TElem * const Y)
    {
        return
            vecadd_axpy_par_alpaka<alpaka::acc::AccGpuCudaRt<alpaka::dim::DimInt<1u>, TSize>, AxpyVectorizedAlpakaKernel>(
                n,
                2u,
                alpha,
                X,
                Y);
    }
    //-----------------------------------------------------------------------------
    //
    //-----------------------------------------------------------------------------
    TReturn vecadd_axpy_par_vec4_alpaka_gpu_cuda(
        TSize const n,
        TElem const alpha,
        TElem const * const X,
        TElem * const Y)
    {
        return
            vecadd_axpy_par_alpaka<alpaka::acc::AccGpuCudaRt<alpaka::dim::DimInt<1u>, TSize>, AxpyVectorizedAlpakaKernel>(
                n,
                4u,
                alpha,
                X,
                Y);
    }
    //-----------------------------------------------------------------------------
    //
    //-----------------------------------------------------------------------------
    TReturn vecadd_axpy_par_vec8_alpaka_gpu_cuda(
        TSize const n,
        TElem const alpha,
        TElem const * const X,
        TElem * const Y)
    {
        return
            vecadd_axpy_par_alpaka<alpaka::acc::AccGpuCudaRt<alpaka::dim::DimInt<1u>, TSize>, AxpyVectorizedAlpakaKernel>(
                n,
                8u,
                alpha,
                X,
                Y);
    }
#endif
