
#include <hip/hip_runtime.h>
//-----------------------------------------------------------------------------
//! \file
//! Copyright 2015 Benjamin Worpitz
//!
//! This file is part of vecadd.
//!
//! vecadd is free software: you can redistribute it and/or modify
//! it under the terms of the GNU Lesser General Public License as published by
//! the Free Software Foundation, either version 3 of the License, or
//! (at your option) any later version.
//!
//! vecadd is distributed in the hope that it will be useful,
//! but WITHOUT ANY WARRANTY; without even the implied warranty of
//! MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See the
//! GNU Lesser General Public License for more details.
//!
//! You should have received a copy of the GNU Lesser General Public License
//! along with vecadd.
//! If not, see <http://www.gnu.org/licenses/>.
//-----------------------------------------------------------------------------

#if defined(VECADD_BUILD_PAR_ALPAKA_ACC_GPU_CUDA)

    #include <vecadd/par/Alpaka.h>

    #include <vecadd/par/Alpaka.hpp>

    //-----------------------------------------------------------------------------
    //
    //-----------------------------------------------------------------------------
    double vecadd_axpy_par_alpaka_gpu_cuda(
        TIdx const n,
        TElem const alpha,
        TElem const * const X,
        TElem * const Y)
    {
        return vecadd_axpy_par_alpaka<alpaka::acc::AccGpuCudaRt<alpaka::dim::DimInt<1u>, TIdx>, VecAddAlpakaKernel>(
            n,
            alpha,
            X,
            Y);
    }
#endif
